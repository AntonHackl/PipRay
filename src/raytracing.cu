#include "hip/hip_runtime.h"
#include <optix_device.h>
#include <optix.h>
#include <hip/hip_runtime.h>
#include "common.h"

extern "C" __constant__ LaunchParams params;

extern "C" __global__ void __raygen__rg()
{
    const float3 orig = params.ray_gen.origin;
    const float3 dir  = params.ray_gen.direction;

    unsigned int p0 = 0;  // hit flag (0 = miss, 1 = hit)
    unsigned int p1 = __float_as_uint(0.0f);

    optixTrace(
        params.handle,
        orig,
        dir,
        0.0f,
        1e16f,
        0.0f,
        OptixVisibilityMask(255),
        OPTIX_RAY_FLAG_NONE,
        /*SBT offset*/ 0,
        /*SBT stride*/ 1,
        /*missSBTIndex*/ 0,
        /*payload*/ p0, p1);

    if (params.result) {
        params.result->hit = p0;
        params.result->t = __uint_as_float(p1);
        
        // Calculate hit point if we hit something
        if (p0) {
            float t = __uint_as_float(p1);
            params.result->hit_point = make_float3(
                orig.x + t * dir.x,
                orig.y + t * dir.y,
                orig.z + t * dir.z
            );
        } else {
            params.result->hit_point = make_float3(0.0f, 0.0f, 0.0f);
        }
    }
}

extern "C" __global__ void __miss__ms()
{
    optixSetPayload_0(0); // hit = 0 (miss)
    optixSetPayload_1(__float_as_uint(0.0f)); // t = 0.0f
}

extern "C" __global__ void __closesthit__ch()
{
    const float2 bc = optixGetTriangleBarycentrics();
    const float t = optixGetRayTmax();
    
    optixSetPayload_0(1); // hit = 1 (hit)
    optixSetPayload_1(__float_as_uint(t)); // distance to hit point
    
    if (params.result) {
        params.result->barycentrics = bc;
    }
} 